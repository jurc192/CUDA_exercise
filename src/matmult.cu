/*
    My very first program in c++
    square a vector
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define ARR_SIZE    64

/* Kernel - square the array */
__global__
void square(float *d_in, float *d_out)
{
    int ndx = threadIdx.x;
    d_out[ndx] = d_in[ndx] * d_in[ndx];
}


int main()
{
    // Allocate array @CPU RAM
    float *h_vec_a = new float[ARR_SIZE];
    float *h_vec_res = new float[ARR_SIZE];

    // Initialize array @CPU
    for (int i=0; i<ARR_SIZE; i++) {
        h_vec_a[i] = (float)i;
    }

    // Allocate arrays @GPU
    float *d_vec_a;
    float *d_vec_res;
    const long arr_bytes = ARR_SIZE * sizeof(float);
    hipMalloc((void **) &d_vec_a, arr_bytes);       // Kako to deluje točno
    hipMalloc((void **) &d_vec_res, arr_bytes);


    // Transfer data from CPU to GPU
    hipMemcpy(d_vec_a, h_vec_a, arr_bytes, hipMemcpyHostToDevice);

    // Kernel launch
    square<<<1, ARR_SIZE>>>(d_vec_a, d_vec_res);

    // Transfer data from GPU to CPU
    hipMemcpy(h_vec_res, d_vec_res, arr_bytes, hipMemcpyDeviceToHost);

    // Delete array
    delete[] h_vec_a;
    delete[] h_vec_res;
    
    hipFree(d_vec_a);
    hipFree(d_vec_res);
    delete d_vec_a;
    delete d_vec_res;

    return 0;
}