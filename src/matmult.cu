/*
    My very first program in c++
    square a vector
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define ARR_SIZE    64

/* Kernel - square the array */
__global__
void square(float *d_in, float *d_out)
{
    int ndx = threadIdx.x;
    d_out[ndx] = d_in[ndx] * d_in[ndx];
}


int main()
{
    // Allocate array @CPU RAM
    float h_vec_a[ARR_SIZE];
    float h_vec_res[ARR_SIZE];

    // Initialize array @CPU
    for (int i=0; i<ARR_SIZE; i++) {
        h_vec_a[i] = (float)i;
        h_vec_res[i] = 0;
    }

    // Allocate arrays @GPU
    float *d_vec_a;
    float *d_vec_res;
    const long arr_bytes = ARR_SIZE * sizeof(float);
    
    if (hipMalloc((void **) &d_vec_a, arr_bytes) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_a\n";
    }
    
    if (hipMalloc((void **) &d_vec_res, arr_bytes) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_res\n";
    }


    
    // Transfer data from CPU to GPU
    hipError_t err;
    err = hipMemcpy(d_vec_a, h_vec_a, arr_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        std::cout << "Failed at hipMemcpy D2H at" << __FILE__ << " line: " << __LINE__ << "\n";
    }

    // Kernel launch
    square<<<1, ARR_SIZE>>>(d_vec_a, d_vec_res);

    // Transfer data from GPU to CPU
    err = hipMemcpy(h_vec_res, d_vec_res, arr_bytes, hipMemcpyDeviceToHost);
    if ( err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        std::cout << "Failed at hipMemcpy D2H at" << __FILE__ << " line: " << __LINE__ << "\n";
    }


    std::cout << "h_vec_res:\n";
    for (int i=0; i< ARR_SIZE; i++) {
        std::cout << h_vec_res[i] << "\n";
    }

    // Free GPU memory
    hipFree(d_vec_a);
    hipFree(d_vec_res);

    // Freeing host memory produces some weird crap -> investigate and/or do c++ style
    
    return 0;
}