#include "hip/hip_runtime.h"
/*
    My very first program in c++
    square a vector
*/
#include <iostream>

#define ARR_SIZE    64

/* Kernel - square the array */
__global__
void square(float *d_in, float *d_out)
{
    int ndx = threadIdx.x;
    d_out[i] = d_in[ndx] * d_in[ndx];
}


int main()
{
    // Allocate array @CPU RAM
    float *h_vec_a = new float[ARR_SIZE];
    float *h_vec_res = new float[ARR_SIZE];

    // Initialize array @CPU
    for (int i=0; i<ARR_SIZE; i++) {
        h_vec_a[i] = (float)i;
    }

    // Allocate arrays @GPU
    float *d_vec_a;
    float *d_vec_res;
    hipMalloc((void **) &d_vec_a, ARR_SIZE * sizeof(float));       // Kako to deluje točno
    hipMalloc((void **) &d_vec_res, ARR_SIZE * sizeof(float));


    // Transfer data from CPU to GPU
    hipMemcpy(d_vec_a, h_vec_a, ARR_SIZE*sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch
    sqare<<<1, ARR_SIZE>>>(d_vec_a, d_vec_res);

    // Transfer data from GPU to CPU
    hipMemcpy(h_vec_res, d_vec_res, ARR_SIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Delete array
    delete[] vec_a;
    delete[] vec_res;

    return 0;
}