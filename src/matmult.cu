/*
    My very first program in c++
    square a vector
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define ARR_SIZE    64

/* Kernel - square the array */
__global__
void square(float *d_in, float *d_out)
{
    int ndx = threadIdx.x;
    d_out[ndx] = d_in[ndx] * d_in[ndx];
}


int main()
{
    // Allocate array @CPU RAM
    float h_vec_a[ARR_SIZE];
    float h_vec_res[ARR_SIZE];

    // Initialize array @CPU
    for (int i=0; i<ARR_SIZE; i++) {
        h_vec_a[i] = (float)i;
        h_vec_res[i] = 0;
    }

    // Allocate arrays @GPU
    float *d_vec_a;
    float *d_vec_res;
    const long arr_bytes = ARR_SIZE * sizeof(float);
    
    if (hipMalloc((void **) &d_vec_a, arr_bytes) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_a\n";
    }
    
    if (hipMalloc((void **) &d_vec_res, arr_bytes) != hipSuccess) {
        std::cout << "Failed at hipMalloc vec_res\n";
    }


    // Transfer data from CPU to GPU
    hipError_t err = hipMemcpy(d_vec_a, h_vec_a, arr_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << "Failed at hipMemcpy H2D\n";
        std::cout << hipGetErrorString(err) << "\n";
    }

    // // Kernel launch
    // square<<<1, ARR_SIZE>>>(d_vec_a, d_vec_res);

    // // Transfer data from GPU to CPU
    // if (cudaMemcpy(h_vec_res, d_vec_res, arr_bytes, cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "Failed at cudaMemcpy D2H\n";
    // }

    // // Delete array
    // delete[] h_vec_a;
    // delete[] h_vec_res;
    
    hipFree(d_vec_a);
    hipFree(d_vec_res);
    // delete d_vec_a;
    // delete d_vec_res;

    return 0;
}